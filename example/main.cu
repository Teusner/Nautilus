#include "core/Scene.cuh"
#include "core/Material.cuh"

#include <iostream>
#include <cmath>

#include "core/Solver.cuh"
#include "core/FrequencyDomain.cuh"
#include "core/kernels.cuh"
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>

#include <hip/hip_runtime_api.h>


int main(void) {
    constexpr unsigned int x = 100;
    constexpr unsigned int y = 100;
    constexpr unsigned int z = 100;

    constexpr float dx = 0.1;
    constexpr float dy = 0.1;
    constexpr float dz = 0.1;

    constexpr float dt = 0.1;

    // FrequencyDomain
    FrequencyDomain fd(2*3.14*2., 2*3.14*25.);
    fd.tau(20);

    Scene s(x, y, z, dx, dy, dz, dt);

    thrust::device_vector<unsigned int> s_M(x * y * z, 0);
    s.SetScene(s_M);
    s.AllocateMaterials(M);

    SinEmitter e(10, 10, 10);
    s.emitters.push_back(e);

    Solver solver;
    hipProfilerStart();
    unsigned int a = 50;
    for (unsigned int i = 0; i < a; i++) {
        solver.Step<x, y, z, SinEmitter>(s);
        s.m_i ++;
    }
    hipProfilerStop();

    // std::cout << "P  : ";
    // thrust::copy(s.P.x.begin() + 1000, s.P.x.begin() + 1010, std::ostream_iterator<float>(std::cout, " "));
    // std::cout << "\nPxy : ";
    // thrust::copy(s.P.xy.begin() + 1000, s.P.xy.begin() + 1010, std::ostream_iterator<float>(std::cout, " "));
    // std::cout << "\nUx  : ";
    // thrust::copy(s.U.x.begin() + 1000, s.U.x.begin() + 1010, std::ostream_iterator<float>(std::cout, " "));
    // std::cout << "\nRx : ";
    // thrust::copy(s.R.x.begin() + 1000, s.R.x.begin() + 1010, std::ostream_iterator<float>(std::cout, " "));
    // std::cout << "\nRxy : ";
    // thrust::copy(s.R.xy.begin() + 1000, s.R.xy.begin() + 1010, std::ostream_iterator<float>(std::cout, " "));
    
    return 0;
}